/**********************************************************************
* FILENAME :        an_kernel.cu             
* 
* DESCRIPTION :
*       Kernel side implementation of AlexNet network
*
* NOTES :
*       This file includes implementation of 2D/3D convolution
*       normalisation,pooling and fully connected layer kernels.
* 
* AUTHOR :    Aajna Karki 
*             https://www.linkedin.com/in/aajna/
*********************************************************************/
#ifndef _AN_KERNEL_H_
#define _AN_KERNEL_H_

#include "hip/hip_runtime.h"
#include <stdio.h>

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) CUT_BANK_CHECKER(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) CUT_BANK_CHECKER(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif


//#define LAYER2_DEBUG 
//#define POOL_DEBUG 
__constant__ int kernelTemplate[25] = {
        0,  1,  2,  3,  4,
        29, 30, 31, 32, 33,
        58, 59, 60, 61, 62,
        87, 88, 89, 90, 91,
        116,117,118,119,120 };
__constant__ int kernelTemplate2[25] = {
        0,  1,  2,  3,  4,
        13, 14, 15, 16, 17, 
        26, 27, 28, 29, 30,
        39, 40, 41, 42, 43, 
        52, 53, 54, 55, 56   };

#ifndef CPU
__global__ void executeFirstLayer(float *bias,float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU,int r_offset, int c_offset)
{
    float product = 0.0;
    int col_width = 227;
    int stride_width = 4;
    int stride = 0,colstride = 0;
    int output = blockIdx.x;
    int row = threadIdx.x + r_offset;
    int col = threadIdx.y + c_offset;
    colstride = 3*row*stride_width*col_width;
    stride = 0;
    product = 0;
    stride = col * 4 * 3;
    /* RGB weights and input 11*11*3 */
    for(int i = 0; i < 11; i++)
    {
        for(int j = 0; j < 11; j++)
        {
            product +=        ((Layer1_Neurons_GPU[i*227*3 + j*3 + stride + colstride]    * Layer1_Weights_GPU[i*11 + j + (output * 11*11*3)])
                    + (Layer1_Neurons_GPU[i*227*3 + j*3 + 1 + stride + colstride] * Layer1_Weights_GPU[i*11 + 11*11 + j+ (output * 11*11*3)])
                    + (Layer1_Neurons_GPU[i*227*3 + j*3 + 2 + stride + colstride] * Layer1_Weights_GPU[i*11 + 11*11*2 + j+ (output * 11*11*3)]));
        }
    }
    product += bias[output];
    if(product < 0) /* RELU Layer */
        product = 0; // max(0,x)
    Layer2_Neurons_GPU[output*55*55 + row*55 + col] = product;
    product = 0.0;
}
/* IN : Layer2_Neurons_GPU // Neurons input
        Layer2_pool_GPU    // output after pooling
        out                // number of outputs 
        out_fr             // feature map size of output in terms of row 
        out_fc             // feature map size of output in terms of column
        kernel             // kernel size
        stride_width       // stride
        in_fr             // feature map size of input in terms of row
        in_fc             // feature map size of input in terms of column 
*/
__global__ void executepoolingCuda(float *Layer2_Neurons_GPU,float *Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    float max = 0.0;
    int stride = 0,colstride = 0;
    int output = blockIdx.x;
    int row = threadIdx.x;
    int col = threadIdx.y;
    colstride = row * stride_width*in_fc;
    stride = col * stride_width;
    for(int i = 0; i < kernel; i++)
    {
        for(int j = 0; j < kernel; j++)
        {
            if(max < ((Layer2_Neurons_GPU[(output*in_fr*in_fc) + i*in_fc + j + stride + colstride])))
                max =   ((Layer2_Neurons_GPU[(output*in_fr*in_fc) + i*in_fc + j + stride + colstride])) ;

        }
    }
    Layer2_pool_GPU[output*out_fr*out_fc + row*out_fc + col] = max;
    max = 0.0;
    stride+= stride_width;
}
__global__ void execute3DconvolutionCuda(float *bias,float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU,int out,int fr,int fc,int stride_width,int kernel,int pad,int in_output,int group)
{
    float product = 0.0;
    int x_pad = 0, y_pad = 0, loopc = 0,loopr = 0;
    int stride = 0,colstride = 0;
    int output = blockIdx.x; // 128
    colstride = 0;
    int row = threadIdx.x;
    stride = 0;
    if(row > pad)
       colstride = (row - pad) * fr;
    int col = threadIdx.y;
    if(col >= pad)
        stride = col * stride_width;
    x_pad = 0; y_pad = 0;
    /* set the loops value */
    loopc = kernel;loopr = kernel;
    /* take care of padding in left hand side of image*/
    if( row < pad)
    {
        x_pad = pad - row;
        loopr = kernel - x_pad;
    }
    /* take care of padding in upper side of image*/
    if( col < pad )
    {
        y_pad = pad - col;
        loopc = kernel - y_pad;
    }
    /* take care of padding in right side of image*/
    if(col >= fc - pad)
        loopc =  fc + pad - col;
    /* take care of padding in bottom of image */
    if(row >= fr - pad)
        loopr =  fr + pad - row;
    for(int feature =0; feature < in_output ; feature++) // calculate the feature maps
    {
        for(int i =0; i < loopr ; i++) // kernel convolution
        {
            for(int j =0; j < loopc ; j++) // kernel convolution
            {
                product += ( Layer2_Neurons_GPU[feature*fr*fc + i*fc + j + stride + colstride] * Layer2_Weights_GPU[output*kernel*kernel*in_output + feature*kernel*kernel + i*kernel + j + kernel*x_pad + y_pad]);
            }
        }
    }
    product += bias[output];
    if(product < 0) /* ReLU Layer */
        product = 0;
    Layer3_Neurons_GPU[output*fr*fc + row*fc + col] = product;
    product = 0.0;
    if(col >= pad)
        stride+=stride_width;
}
__global__ void execute3Dconvolutiongroup2Cuda(float *bias,float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU,int out,int fr,int fc,int stride_width,int kernel,int pad,int in_output,int group)
{
    float product = 0.0;
    int x_pad = 0, y_pad = 0, loopc = 0,loopr = 0;
    int stride = 0,colstride = 0;
    /* Execute second set of inputs */
    int output = blockIdx.x + out;
    colstride = 0;
    int row = threadIdx.x;
    stride = 0;
    if(row > pad)
        colstride = (row - pad) * fr;
    int col = threadIdx.y;
    if(col >= pad)
        stride = col*stride_width;
    x_pad = 0; y_pad = 0;
    /* set the loops value */
    loopc = kernel;loopr = kernel;
    /* take care of padding in left hand side of image*/
    if( row < pad)
    {
        x_pad = pad - row;
        loopr = kernel - x_pad;
    }
    /* take care of padding in upper side of image*/
    if( col < pad )
    {
        y_pad = pad - col;
        loopc = kernel - y_pad;
    }
    /* take care of padding in right side of image*/
    if(col >= fc - pad)
        loopc =  fc + pad - col;
    /* take care of padding in bottom of image */
    if(row >= fr - pad)
        loopr =  fr + pad - row;
    for(int feature = in_output ; feature < (in_output << 1) ; feature++) // calculate the feature maps
    {
        for(int i =0; i < loopr ; i++) // kernel convolution
        {
            for(int j =0; j < loopc ; j++) // kernel convolution
            {
                product += (( Layer2_Neurons_GPU[feature*fr*fc + i*fc + j + stride + colstride] * Layer2_Weights_GPU[output*kernel*kernel*in_output + (feature-in_output)*kernel*kernel + i*kernel + j + kernel*x_pad + y_pad]));
            }
        }
    }
    product += bias[output];
    if(product < 0) /* ReLU Layer */
        product = 0;
    Layer3_Neurons_GPU[output*fr*fc + row*fc + col] = product;
    product = 0.0;
}
__global__ void executelrnNormCuda_split(float *Layer_InNeurons_GPU, float alpha, float beta,int local_size,int out,int fr,int fc,float *Layer_OutNeurons_GPU,int r_offset, int c_offset)
{
        int nStart = 0, nEnd = 0;
        float value = 0.0;float sum = 0.0;
        int output = blockIdx.x;
        int row = threadIdx.x + r_offset;
        int col = threadIdx.y + c_offset;
        nStart=(output-2) > 1 ? (output-2) : 1 ;
        nEnd=(output+2) <  out ? (output+2) : out ;
        for(int i = (nStart-1); i < (nEnd-1) ; i++) // kernel convolution
        {
            sum += pow(( Layer_InNeurons_GPU[i*fr*fc + row*fc + col]),2);
        }
        value = (Layer_InNeurons_GPU[output*fr*fc + row*fc + col]) / (pow( 1 + ((alpha/local_size) *sum),beta));
        sum = 0;
        Layer_OutNeurons_GPU[output*fr*fc + row*fc + col] = value;
}
__global__ void executelrnNormCuda(float *Layer_InNeurons_GPU, float alpha, float beta,int local_size,int out,int fr,int fc,float *Layer_OutNeurons_GPU,int func_call)
{
        int nStart = 0, nEnd = 0;
        float value = 0.0;float sum = 0.0;
        int output = blockIdx.x;
        int row = threadIdx.x + func_call * 32;
        int col = threadIdx.y + func_call * 32;
        nStart=(output-2) > 1 ? (output-2) : 1 ;
        nEnd=(output+2) <  out ? (output+2) : out ;
        for(int i = (nStart-1); i < (nEnd-1) ; i++) // kernel convolution
        {
            sum += pow(( Layer_InNeurons_GPU[i*fr*fc + row*fc + col]),2);
        }
        value = (Layer_InNeurons_GPU[output*fr*fc + row*fc + col]) / (pow( 1 + ((alpha/local_size) *sum),beta));
        sum = 0;
        Layer_OutNeurons_GPU[output*fr*fc + row*fc + col] = value;
}
__global__ void executeFCLayer(float *bias,float *Layer_InNeurons_GPU,float *Layer_Weights_GPU,float *Layer_OutNeurons_GPU,int output, int input,bool reLU,bool dropout)
{
    float product = 0.0;
    int out = blockIdx.x;
    int weight = out * input;
    {
        for(int in = 0; in < input; in++)
        {
               product += Layer_InNeurons_GPU[in] * Layer_Weights_GPU[weight+in];
        }
        product += bias[out];
        if(reLU == true)
        {
            if(product < 0) /* ReLU Layer */
                product = 0;
        }

        Layer_OutNeurons_GPU[out] = product;
        product = 0.0;
    }
}
__global__ void executeThirdLayer(float *Layer3_Neurons_GPU, float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU)
{
    int blockID=blockIdx.x;
    //int pixelY=threadIdx.y;


    int weightBegin=blockID*1251;

    float result=0;

    result+=Layer3_Weights_GPU[weightBegin];

    ++weightBegin;

    for (int i=0; i<1250; ++i )
    {
        result+=Layer3_Neurons_GPU[i+(1250*blockIdx.y)]*Layer3_Weights_GPU[weightBegin+i];
    }

    result=(1.7159*tanhf(0.66666667*result));

    Layer4_Neurons_GPU[blockID+(100*blockIdx.y)]=result;

}

__global__ void executeFourthLayer(float *Layer4_Neurons_GPU,float *Layer4_Weights_GPU,float *Layer5_Neurons_GPU)
{
    int blockID=blockIdx.x;
    //int pixelY=threadIdx.y;


    int weightBegin=blockID*101;

    float result=0;

    result+=Layer4_Weights_GPU[weightBegin];

    ++weightBegin;

    for (int i=0; i<100; ++i )
    {
        result+=Layer4_Neurons_GPU[i+(100*blockIdx.y)]*Layer4_Weights_GPU[weightBegin+i];
    }

    result=(1.7159*tanhf(0.66666667*result));

    Layer5_Neurons_GPU[blockID+(10*blockIdx.y)]=result;
}
#else
void executeFirstLayer(float *bias,float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU,int stride_width,int col_width,int feature_r,int feature_c,int out)
{
    float product = 0.0;
    int stride = 0,colstride = 0;
    {
        for(int output =0;output < out ;output++)
        {
            for(int row =0; row < feature_r ;row++)
            {
                                colstride = 3*row*stride_width*col_width;
                stride = 0;
                for(int col =0; col < feature_c ;col++)
                {
                    product = 0;
                    /* RGB weights and input 11*11*3 , kernel is 11*11 */
                    for(int i = 0; i < 11; i++)
                    {
                        for(int j = 0; j < 11; j++)
                        {
                            product +=        ((Layer1_Neurons_GPU[i*col_width*3 + j*3 + stride + colstride]    * Layer1_Weights_GPU[i*11 + j + (output * 11*11*3)])
                                    + (Layer1_Neurons_GPU[i*col_width*3 + j*3 + 1 + stride + colstride] * Layer1_Weights_GPU[i*11 + 11*11 + j+ (output * 11*11*3)])
                                    + (Layer1_Neurons_GPU[i*col_width*3 + j*3 + 2 + stride + colstride] * Layer1_Weights_GPU[i*11 + 11*11*2 + j+ (output * 11*11*3)]));
                        }
                    }
                    product += bias[output];
                                        if(product < 0) /* RELU Layer */
                        product = 0; // max(0,x)
                    Layer2_Neurons_GPU[output*feature_r*feature_c + row*feature_c + col] = product;
#ifdef LAYER1_DEBUG         
                    printf("%f\n",product);
#endif
                    product = 0.0;
                    stride+= stride_width*3;
                }
            }
        }
    }
}
void pooling(float *Layer2_Neurons_GPU,float *Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
        printf("pooling Activation layer \n");
    float max = 0.0;
    int downsample = 0;
    int stride = 0,colstride = 0;
    {
        for(int output =0;output < out ;output++)
        {
            for(int row =0; row < out_fr ;row++)
            {
                                colstride = row * stride_width*in_fc;
                stride = 0;
                for(int col =0; col < out_fc ;col++)
                {
                    for(int i = 0; i < kernel; i++)
                    {
                        for(int j = 0; j < kernel; j++)
                        {
                            if(max < ((Layer2_Neurons_GPU[(output*in_fr*in_fc) + i*in_fc + j + stride + colstride])))
                                max =   ((Layer2_Neurons_GPU[(output*in_fr*in_fc) + i*in_fc + j + stride + colstride])) ;
                //          if(output == 141)
                //              printf("%f %d\t",Layer2_Neurons_GPU[(output*in_fr*in_fc) + i*in_fc + j + stride + colstride],((output*in_fr*in_fc) + i*in_fc + j + stride + colstride)) ;

                        }
                    }
                    Layer2_pool_GPU[downsample] = max;
#ifdef POOL_DEBUG           
                    printf("\n %f %d\n",max,downsample);
#endif
                    max = 0.0;
                    downsample++;
                    stride+= stride_width;
                }
            }
        }
    }
}

void execute3Dconvolution(float *bias,float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU,int out,int fr,int fc,int stride_width,int kernel,int pad,int in_output,int group)
{
	float product = 0.0;
        int x_pad = 0, y_pad = 0, loopc = 0,loopr = 0;
        printf(" 3D convolution with group %d,output %d,feature %d x %d ,stride %d, kernel %d, pad %d, input %d\n",group,out,fr,fc,stride_width,kernel,pad,in_output); 
	if(group == 2)
	{
		out = out >> 1;
		in_output = in_output >> 1;
	}
	int stride = 0,colstride = 0;
	{
		for(int output =0;output < out  ;output++) /* out = 256 */
		{      
                        colstride = 0;	
			for(int row =0; row < fr ; row++) /* out = 256 */
			{	
				stride = 0;	
				if(row > pad)
					colstride = (row - pad) * fr;
				for(int col =0; col < fc ;col++) /* out = 256 */
				{
					x_pad = 0; y_pad = 0;
					/* set the loops value */
					loopc = kernel;loopr = kernel;
					/* take care of padding in left hand side of image*/ 
					if( row < pad)
					{
						x_pad = pad - row;
						loopr = kernel - x_pad;
					} 
					/* take care of padding in upper side of image*/ 
					if( col < pad )
					{
						y_pad = pad - col;
						loopc = kernel - y_pad;
					} 
					/* take care of padding in right side of image*/ 
					if(col >= fc - pad)
						loopc =  fc + pad - col;  
					/* take care of padding in bottom of image */ 
					if(row >= fr - pad)
						loopr =  fr + pad - row;
					for(int feature =0; feature < in_output ; feature++) // calculate the feature maps
					{
						for(int i =0; i < loopr ; i++) // kernel convolution
						{
							for(int j =0; j < loopc ; j++) // kernel convolution
							{
								product += ( Layer2_Neurons_GPU[feature*fr*fc + i*fc + j + stride + colstride] * Layer2_Weights_GPU[output*kernel*kernel*in_output + feature*kernel*kernel + i*kernel + j + kernel*x_pad + y_pad]);
							}
						}
					}
                                        product += bias[output];
					if(product < 0) /* ReLU Layer */
						product = 0;
#ifdef LAYER2_DEBUG
					printf("%f\n",product);
#endif                  
//					if((group == 2) && (out == 128) && (in_output == 192))
//						printf("%f\n",product);
					Layer3_Neurons_GPU[output*fr*fc + row*fc + col] = product;
					product = 0.0;
					if(col >= pad)
						stride+=stride_width;
				}
			}

		}
		if(group == 2)
		{
			/* Execute second set of inputs */
			for(int output = out ;output < (out << 1)   ;output++) /* out = 256 */
			{      
				colstride = 0;	
				for(int row =0; row < fr; row++) /* out = 256 */
				{	
					stride = 0;	
					if(row > pad)
						colstride = (row - pad) * fr;
					for(int col =0; col < fc ;col++) /* out = 256 */
					{
						x_pad = 0; y_pad = 0;
						/* set the loops value */
						loopc = kernel;loopr = kernel;
						/* take care of padding in left hand side of image*/ 
						if( row < pad)
						{
							x_pad = pad - row;
							loopr = kernel - x_pad;
						} 
						/* take care of padding in upper side of image*/ 
						if( col < pad )
						{
							y_pad = pad - col;
							loopc = kernel - y_pad;
						} 
						/* take care of padding in right side of image*/ 
						if(col >= fc - pad)
							loopc =  fc + pad - col;  
						/* take care of padding in bottom of image */ 
						if(row >= fr - pad)
							loopr =  fr + pad - row;
						for(int feature = in_output ; feature < (in_output << 1) ; feature++) // calculate the feature maps
						{
							for(int i =0; i < loopr ; i++) // kernel convolution
							{
								for(int j =0; j < loopc ; j++) // kernel convolution
								{
									product += (( Layer2_Neurons_GPU[feature*fr*fc + i*fc + j + stride + colstride] * Layer2_Weights_GPU[output*kernel*kernel*in_output + (feature-in_output)*kernel*kernel + i*kernel + j + kernel*x_pad + y_pad]));
								}
							}
						}
						product += bias[output];
						if(product < 0) /* ReLU Layer */
							product = 0;
#ifdef LAYER2_DEBUG
						printf("%f\n",product);
#endif                   
//						if((group == 2) && (out == 128) && (in_output == 192))
//							printf("%f\n",product);
						Layer3_Neurons_GPU[output*fr*fc + row*fc + col] = product;
						product = 0.0;
						if(col >= pad)
							stride+=stride_width;
					}
				}

			}
		}

	}
}
void executelrnNorm(float *Layer_InNeurons_GPU, float alpha, float beta,int local_size,int out,int fr,int fc,float *Layer_OutNeurons_GPU)
{
        printf(" Exexcute Norm Layer\n");
        int nStart = 0, nEnd = 0;
        float value = 0.0;float sum = 0.0;
	for(int row =0; row < fr; row++) 
	{	
		for(int col =0; col < fc ;col++)
		{
			for(int output = 0 ;output < out   ;output++)
			{
                                nStart=(output-floor(local_size/2)) > 1 ? (output-floor(local_size/2)) : 1 ;
                                nEnd=(output+floor(local_size/2)) <  out ? (output+floor(local_size/2)) : out ;
				for(int i = (nStart-1); i < (nEnd-1) ; i++) // kernel convolution
				{
					sum += pow(( Layer_InNeurons_GPU[i*fr*fc + row*fc + col]),2);
				}
				value = (Layer_InNeurons_GPU[output*fr*fc + row*fc + col]) / (pow( 1 + ((alpha/local_size) *sum),beta));
                                sum = 0;
                                Layer_OutNeurons_GPU[output*fr*fc + row*fc + col] = value;
			}
		}

	}
#ifdef NORM_LAYER
	for(int N = 0; N < out; N++)
	{

		for(int W = 0; W < fr; W++)
		{
			for(int H = 0; H < fc; H++)
			{
                                printf("%f\n",Layer_OutNeurons_GPU[N*fr*fc + W*fc + H]);;
			}
		}
	}
#endif
}
void executeFCLayer(float *bias,float *Layer_InNeurons_GPU,float *Layer_Weights_GPU,float *Layer_OutNeurons_GPU,int output, int input,bool reLU,bool dropout)
{
        printf("Execute FC Layer of output : %d input %d\n",output,input);
        float product = 0.0,max = 0.0; int weight = 0,index = 0;
	for(int out=0; out < output ; out++)
	{
		for(int in = 0; in < input; in++)
		{
                     product += Layer_InNeurons_GPU[in] * Layer_Weights_GPU[weight++];
		}
                product += bias[out];
		if(reLU == true)
		{
			if(product < 0) /* ReLU Layer */
				product = 0;
		}
		else
		{
			if(max < product)
			{
                                index = out;
				max = product;
			}
		}
                if(dropout == true)
		{

		}
		Layer_OutNeurons_GPU[out] = product;
#ifdef FC_DEBUG
		printf("%f\n",product);
#endif
		product = 0.0;
	}
        printf(" MAX from FC layer = %d\n",index);
}

void executeSoftMax(float *Layer_In_Neurons_GPU)
{
        printf("executeSoftMax \n");
        float max = 0,sum = 0;
	float output[1000] = {0};
        for(int i = 0; i < 1000; i++)
	{
		if(Layer_In_Neurons_GPU[i] > max)
			max = Layer_In_Neurons_GPU[i];
	}
#ifdef SOFTMAX_DEBUG
	printf("Max = %10e\n",max);
#endif
        for(int i = 0; i < 1000; i++)
	{
		output[i] = exp(Layer_In_Neurons_GPU[i] - max);
                sum += output[i];
	}
#ifdef SOFTMAX_DEBUG
	printf("Sum =  %10e\n",sum);
#endif
        for(int i = 0; i < 1000; i++)
	{
		output[i] *= (1/sum);
#ifdef SOFTMAX_DEBUG
                printf("%10e\n",output[i]);
#endif
	}
        
}
#endif


#endif // #ifndef _AN_KERNEL_H_
