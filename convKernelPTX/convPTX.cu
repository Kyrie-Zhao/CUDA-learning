#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>
#include <time.h>

#define BLOCK_SIZE 32
#define WA 64 
#define HA 64   
#define HC 3     
#define WC 3
#define WB (WA - WC + 1)
#define HB (HA - HC + 1)


__global__ void Convolution(float* A, float* B, float* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
	int col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
	int row = blockIdx.y * (BLOCK_SIZE - WC + 1) + threadIdx.y;
	int row_i = row - WC + 1;
	int col_i = col - WC + 1;

	float tmp = 0;

	__shared__ float shm[BLOCK_SIZE][BLOCK_SIZE];

	if (row_i < WA && row_i >= 0 && col_i < WA && col_i >= 0)
	{
		shm[threadIdx.y][threadIdx.x] = A[col_i * WA + row_i];
	}
	else
	{
		shm[threadIdx.y][threadIdx.x] = 0;
	}

	__syncthreads();

	if (threadIdx.y < (BLOCK_SIZE - WC + 1) && threadIdx.x < (BLOCK_SIZE - WC + 1) && row < (WB - WC + 1) && col < (WB - WC + 1))
	{
		for (int i = 0; i< WC;i++)
			for (int j = 0;j<WC;j++)
				tmp += shm[threadIdx.y + i][threadIdx.x + j] * C[j*WC + i];
		B[col*WB + row] = tmp;
	}
}