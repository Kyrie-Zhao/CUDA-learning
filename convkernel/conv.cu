#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <time.h>

#define BLOCK_SIZE 32
#define WA 64 
#define HA 64   
#define HC 3     
#define WC 3
#define WB (WA - WC + 1)
#define HB (HA - HC + 1)


__global__ void Convolution(float* A, float* B, float* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
	int col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
	int row = blockIdx.y * (BLOCK_SIZE - WC + 1) + threadIdx.y;
	int row_i = row - WC + 1;
	int col_i = col - WC + 1;

	float tmp = 0;

	__shared__ float shm[BLOCK_SIZE][BLOCK_SIZE];

	if (row_i < WA && row_i >= 0 && col_i < WA && col_i >= 0)
	{
		shm[threadIdx.y][threadIdx.x] = A[col_i * WA + row_i];
	}
	else
	{
		shm[threadIdx.y][threadIdx.x] = 0;
	}

	__syncthreads();

	if (threadIdx.y < (BLOCK_SIZE - WC + 1) && threadIdx.x < (BLOCK_SIZE - WC + 1) && row < (WB - WC + 1) && col < (WB - WC + 1))
	{
		for (int i = 0; i< WC;i++)
			for (int j = 0;j<WC;j++)
				tmp += shm[threadIdx.y + i][threadIdx.x + j] * C[j*WC + i];
		B[col*WB + row] = tmp;
	}
}


void randomInit(float* data, int size)
{
	for (int i = 0; i < size; ++i)
		data[i] = rand() / (float)RAND_MAX;
    
}

int main(int argc, char** argv)
{
	srand(2006);
	hipError_t error;
	hipEvent_t start_G, stop_G;

	hipEventCreate(&start_G);
	hipEventCreate(&stop_G);

	unsigned int size_A = WA * HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);

	unsigned int size_B = WB * HB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);

	unsigned int size_C = WC * HC;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C = (float*)malloc(mem_size_C);

	randomInit(h_A, size_A);
	randomInit(h_C, size_C);
    for(int loop = 0; loop < size_A; loop++){
        printf("%f ", h_A[loop]);
    }
	float* d_A;
	float* d_B;
	float* d_C;

	error = hipMalloc((void**)&d_A, mem_size_A);
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipMalloc for A\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**)&d_B, mem_size_B);//results
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipMalloc for B\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**)&d_C, mem_size_C);
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipMalloc for C\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}


	error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipMemcpy for A\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	error = hipMemcpy(d_C, h_C, mem_size_C, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipMemcpy for C\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((WB - 1) / (BLOCK_SIZE - WC + 1), (WB - 1) / (BLOCK_SIZE - WC + 1));

	Convolution << < grid, threads >> >(d_A, d_B, d_C, HA, WA, HB, WB, HC, WC);

	hipEventRecord(start_G);

	Convolution << < grid, threads >> >(d_A, d_B, d_C, HA, WA, HB, WB, HC, WC);
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in launching kernel\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	error = hipDeviceSynchronize();

	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipDeviceSynchronize \n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}

	hipEventRecord(stop_G);

	hipEventSynchronize(stop_G);

	error = hipMemcpy(h_B, d_B, mem_size_B, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s  in hipMemcpy for B\n", hipGetErrorString(error));
		return EXIT_FAILURE;
	}


	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start_G, stop_G);

	printf("Time took to compute matrix A of dimensions %d x %d  on GPU is %f ms \n \n \n", WA, HA, miliseconds);

	for (int i = 0;i < HB;i++)
	{
		for (int j = 0;j < WB;j++)
		{
            int i = 1;
			printf("%f ", h_B[i*HB + j]);
		}
		//printf("\n");
	}

	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return EXIT_SUCCESS;
}