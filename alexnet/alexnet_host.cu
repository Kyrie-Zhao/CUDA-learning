#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
// includes, project
//#include <cutil.h>
#define L1_KERNEL_SIZE 11*11*3
#define L1_OUT 96
#define L2_KERNEL_SIZE 5*5*48
#define L2_OUT 256 
#define L3_KERNEL_SIZE 3*3*256
#define L3_OUT 384 
#define L4_KERNEL_SIZE 3*3*192
#define L4_OUT 384
#define L5_KERNEL_SIZE 3*3*192
#define L5_OUT 256
#define INPUT_SIZE 227*227*3

#define L1_FMAP 55*55
#define L2_FMAP 27*27
#define L3_FMAP 13*13
#define L4_FMAP 13*13
#define L5_FMAP 13*13
#define POOL1_FMAP 27*27
#define POOL2_FMAP 13*13
#define POOL3_FMAP 6*6
//#define CPU
//#define NUM 10
// includes, kernels
#include "alexnet_device.cu"
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
extern "C"
void NeuralNetwork();
unsigned g_verbose;
unsigned NUM;
void extract_weights(const char *pFileName,float *layer_weights,bool bias)
{
	FILE * pFile1 = fopen (pFileName,"rb");
	char delim[2];
	if(bias == true)
		delim[0] = ' ';
	else
		delim[0] = '\n';
	delim[1] = 0;
	char *token;
	int count = 0;
	char *line = NULL;
	size_t len = 0;
	if (!(pFile1 != NULL))
            printf("File Not Found\n");
	if (pFile1 != NULL && (bias == false))
	{
		printf(" File FOUND %s\n",pFileName);
		{
			
			//fread(weights,sizeof(weights),1,pFile1);
			//token = strtok(weights,delim);
			//while(token != NULL)
			while (getline(&line, &len, pFile1) != -1)
			{
				token = strtok(line,delim);
				float temp_num = atof(token);
			        layer_weights[count] = temp_num;	
				//printf("%.8f\t",temp_num); 
				count++; 
			//	token = strtok(NULL,delim);
			}
		}
		printf("Final Count : %d\n",count);
		fclose(pFile1);
	}
	if (pFile1 != NULL && (bias == true))
	{
		printf(" File FOUND %s\n",pFileName);
		{

			char weights[94590] = "";
			fread(weights,sizeof(weights),1,pFile1);
			token = strtok(weights,delim);
			while(token != NULL)
			{
				float temp_num = atof(token);
			        layer_weights[count] = temp_num;	
				//printf("%.8f\t",temp_num); 
				count++; 
				token = strtok(NULL,delim);
			}
		}
		printf("Final Count : %d\n",count);
		fclose(pFile1);
	}

}
int main(int argc, char** argv)
{
	int i, commandline_error;
	commandline_error = 0;
	g_verbose = 0;
	if (argc >= 2) {
		NUM = atoi(argv[1]);
		for (i=2; i < argc;i++) {
			if (argv[i][0] == '-') {
				switch (argv[i][1]) {
				case 'v': g_verbose = 1;
					break;
				default: commandline_error=1;
				}
			}
			else commandline_error=1;
		}
	} else commandline_error=1;
	if (commandline_error || !NUM) {
		printf("Usage: ./AN <NUM> [-v]\n");
		printf("where NUM is the number of images to process in parallel (up to 10000 for the t10k-images-idx3-ubyte database file) and -v is used to display approximately what each image looks like.\n");
		return 1;
	}
	NeuralNetwork();
}
void Fill_weights(float *Layer1_Weights_CPU,float *Layer2_Weights_CPU,float *Layer3_Weights_CPU,float *Layer4_Weights_CPU,float *Layer5_Weights_CPU,float *Layer6_Weights_CPU,float *Layer7_Weights_CPU,float *Layer8_Weights_CPU)
{
	extract_weights("data/conv1.txt",Layer1_Weights_CPU,false);
	extract_weights("data/conv2.txt",Layer2_Weights_CPU,false);
	extract_weights("data/conv3.txt",Layer3_Weights_CPU,false);
	extract_weights("data/conv4.txt",Layer4_Weights_CPU,false);
	extract_weights("data/conv5.txt",Layer5_Weights_CPU,false);
	extract_weights("data/fc6.txt",Layer6_Weights_CPU,false);
	extract_weights("data/fc7.txt",Layer7_Weights_CPU,false);
	extract_weights("data/fc8.txt",Layer8_Weights_CPU,false);
	printf("Extracted Weights and Bias successfully\n");
}
void Fill_bias(float *bias_1,float *bias_2,float *bias_3,float *bias_4,float *bias_5,float *bias_6,float *bias_7,float *bias_8)
{
	extract_weights("data/bias1.txt",bias_1,true);
	extract_weights("data/bias2.txt",bias_2,true);
	extract_weights("data/bias3.txt",bias_3,true);
	extract_weights("data/bias4.txt",bias_4,true);
	extract_weights("data/bias5.txt",bias_5,true);
	extract_weights("data/bias6.txt",bias_6,true);
	extract_weights("data/bias7.txt",bias_7,true);
	extract_weights("data/bias8.txt",bias_8,true);
}
void readIn(float *layer1)
{
	FILE *fp = fopen ("data/input.txt","rb");
	size_t len;
        char delim[1];
        delim[0] = '\n';
	int count = 0;
	char *token;
        char *line = NULL;
	if (fp != NULL)
	{
		printf(" File FOUND\n");
		{
			while ((getline(&line, &len, fp)) != -1)
			{
                                token = strtok(line,delim);
                                layer1[count] = atof(token);
				count++;		
			}
			printf("READ INPUT Final Count :: %d\n",count);		
		}
		fclose(fp);
	}
	else
	{
		printf(" File NOt FOUND\n");
	}
}
void NeuralNetwork()
{
#ifndef CPU// Set the CUDA device	
	int deviceCount;                                                         
	hipGetDeviceCount(&deviceCount);                
	if (deviceCount == 0) {                                                  
		fprintf(stderr, "There is no device.\n");                            
		exit(EXIT_FAILURE);                                                  
	}                                                                        
	int dev;                                                                 
	for (dev = 0; dev < deviceCount; ++dev) {                                
		hipDeviceProp_t deviceProp;                                           
		hipGetDeviceProperties(&deviceProp, dev);   
		if (deviceProp.major >= 1)                                           
			break;                                                           
	}                                                                        
	if (dev == deviceCount) {                                                
		fprintf(stderr, "There is no device supporting CUDA.\n");            
		exit(EXIT_FAILURE);                                                  
	}                                                                        
	else                                                                     
		hipSetDevice(dev);
#endif  
        /* Read Input File 227*227*3 */	
	float *Layer1_Neurons_CPU = (float*) malloc (INPUT_SIZE * sizeof(float));
	readIn(Layer1_Neurons_CPU);

        /* Declaration of Bias and Weights for CPU */ 
	float bias_1[96],bias_2[256],bias_3[384],bias_4[384],bias_5[256],bias_6[4096],bias_7[4096],bias_8[1000];
	float *Layer1_Weights_CPU = (float *)malloc(sizeof(float) *(L1_KERNEL_SIZE * L1_OUT));
	float *Layer2_Weights_CPU = (float *)malloc(sizeof(float) *(L2_KERNEL_SIZE * L2_OUT));
	float *Layer3_Weights_CPU = (float *)malloc(sizeof(float) *(L3_KERNEL_SIZE * L3_OUT));
	float *Layer4_Weights_CPU = (float *)malloc(sizeof(float) *(L4_KERNEL_SIZE * L4_OUT));
	float *Layer5_Weights_CPU = (float *)malloc(sizeof(float) *(L5_KERNEL_SIZE * L5_OUT));
	float *Layer6_Weights_CPU = (float *)malloc(sizeof(float) *(4096*256*6*6));
	float *Layer7_Weights_CPU = (float *)malloc(sizeof(float) *(4096*4096));
	float *Layer8_Weights_CPU = (float *)malloc(sizeof(float) *(4096*1000));
        
	/* Fill Bias and Weights */	
	Fill_bias(bias_1,bias_2,bias_3,bias_4,bias_5,bias_6,bias_7,bias_8);
        Fill_weights(Layer1_Weights_CPU,Layer2_Weights_CPU,Layer3_Weights_CPU,Layer4_Weights_CPU,Layer5_Weights_CPU,Layer6_Weights_CPU,Layer7_Weights_CPU,Layer8_Weights_CPU);

        /* if CPU = 1 then CPU version of code ,else CUDA code */
#ifdef CPU 
	/* First Layer convolution + ReLU + pooling */ 
    float *Layer2_Neurons_CPU = (float *)malloc(sizeof(float) *(96*55*55));
	executeFirstLayer(bias_1,Layer1_Neurons_CPU,Layer1_Weights_CPU,Layer2_Neurons_CPU,4,227,55,55,96);
	/*Normalisation */	
    float *Layer2_Norm_CPU = (float *)malloc(sizeof(float) *(96*55*55));
    executelrnNorm(Layer2_Neurons_CPU,0.0001,0.75,5,96,55,55,Layer2_Norm_CPU);
    /* Max Pool */
    float *Layer2_pool_CPU = (float *)malloc(sizeof(float) *(96*27*27));
    pooling(Layer2_Norm_CPU,Layer2_pool_CPU,96,27,27,3,2,55,55);

	/* Second Layer */	
	float *Layer3_Neurons_CPU = (float *)malloc(sizeof(float) *(256*27*27));
	execute3Dconvolution(bias_2,Layer2_pool_CPU,Layer2_Weights_CPU,Layer3_Neurons_CPU,256,27,27,1,5,2,96,2);
    /*Normalisation */
    float *Layer3_Norm_CPU = (float *)malloc(sizeof(float) *(256*27*27));
    executelrnNorm(Layer3_Neurons_CPU,0.0001,0.75,5,256,27,27,Layer3_Norm_CPU);
    /* Max Pool */ 
	float *Layer3_pool_CPU = (float *)malloc(sizeof(float) *(256*13*13));
    pooling(Layer3_Norm_CPU,Layer3_pool_CPU,256,13,13,3,2,27,27);

	/* Third Layer convolution + ReLU  */ 
	float *Layer4_Neurons_CPU = (float *)malloc(sizeof(float) *(384*13*13));
	execute3Dconvolution(bias_3,Layer3_pool_CPU,Layer3_Weights_CPU,Layer4_Neurons_CPU,384,13,13,1,3,1,256,1);
	
	/* Fourth Layer convolution + ReLU  */ 
	float *Layer5_Neurons_CPU = (float *)malloc(sizeof(float) *(384*13*13));
	execute3Dconvolution(bias_4,Layer4_Neurons_CPU,Layer4_Weights_CPU,Layer5_Neurons_CPU,384,13,13,1,3,1,384,2);

	/* Fifth Layer convolution + ReLU + pooling */ 
	float *fc6_Neurons_CPU = (float *)malloc(sizeof(float) *(256*13*13));
	execute3Dconvolution(bias_5,Layer5_Neurons_CPU,Layer5_Weights_CPU,fc6_Neurons_CPU,256,13,13,1,3,1,384,2);
	float *fc6_pool_CPU = (float *)malloc(sizeof(float) *(256*6*6));
    pooling(fc6_Neurons_CPU,fc6_pool_CPU,256,6,6,3,2,13,13);

    /* Sixth Layer Fully connected + ReLU */	
	float *fc7_Neurons_CPU = (float *)malloc(sizeof(float) * (4096));
	executeFCLayer(bias_6,fc6_pool_CPU,Layer6_Weights_CPU,fc7_Neurons_CPU,4096,(256*6*6),true,true);

	/* Seventh Layer Fully connected + ReLU */	
	float *fc8_Neurons_CPU = (float *)malloc(sizeof(float) * (4096));
	executeFCLayer(bias_7,fc7_Neurons_CPU,Layer7_Weights_CPU,fc8_Neurons_CPU,4096,4096,true,true);

    /*Eigth Layer */
	float *fc9_Neurons_CPU = (float *)malloc(sizeof(float) * (1000));
	executeFCLayer(bias_8,fc8_Neurons_CPU,Layer8_Weights_CPU,fc9_Neurons_CPU,1000,4096,false,false);
#else
    /*Layer1 */
    // Layer1 Neurons -> Layer1_norm -> Layer1_pool -> Layer2_Neurons-> 
	float *Layer1_bias_GPU,*Layer1_Weights_GPU,*Layer1_Neurons_GPU,*Layer1_Norm_GPU,*Layer1_pool_GPU,*Layer2_Neurons_GPU;

	hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(float)* INPUT_SIZE);
	hipMalloc((void**) &Layer1_Weights_GPU, sizeof(float)* L1_KERNEL_SIZE * L1_OUT);
	hipMalloc((void**) &Layer1_bias_GPU, sizeof(float)* L1_OUT);
	hipMalloc((void**) &Layer1_Norm_GPU, sizeof(float)* (L1_OUT * L1_FMAP));

	hipMemcpy(Layer1_Weights_GPU,Layer1_Weights_CPU, sizeof(float)*L1_KERNEL_SIZE * L1_OUT, hipMemcpyHostToDevice);
	hipMemcpy(Layer1_Neurons_GPU,Layer1_Neurons_CPU, sizeof(float)*INPUT_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(Layer1_bias_GPU,bias_1, sizeof(float)* L1_OUT, hipMemcpyHostToDevice);
    /* Output is 96*55*55 , hence launch as 96*32*32 + 96*23*23 */
	dim3 Layer1_Block(96,1,1);
	dim3 Layer1_Thread(32,32);	
	executeFirstLayer<<<Layer1_Block,Layer1_Thread>>>(Layer1_bias_GPU,Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer1_Norm_GPU,0,0);
	dim3 Layer11_Block(96,1,1);
	dim3 Layer11_Thread(32,23);	
	executeFirstLayer<<<Layer11_Block,Layer11_Thread>>>(Layer1_bias_GPU,Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer1_Norm_GPU,0,32);
	dim3 Layer12_Block(96,1,1);
	dim3 Layer12_Thread(23,32);	
	executeFirstLayer<<<Layer12_Block,Layer12_Thread>>>(Layer1_bias_GPU,Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer1_Norm_GPU,32,0);
	dim3 Layer13_Block(96,1,1);
	dim3 Layer13_Thread(23,23);	
	executeFirstLayer<<<Layer13_Block,Layer13_Thread>>>(Layer1_bias_GPU,Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer1_Norm_GPU,32,32);

	/*Normalisation */	
	hipMalloc((void**) &Layer1_pool_GPU,sizeof(float)* L1_OUT*L1_FMAP);
	dim3 Norm1_Block(96,1,1);
	dim3 Norm1_Thread(32,32);   
	executelrnNormCuda_split<<<Norm1_Block,Norm1_Thread>>>(Layer1_Norm_GPU,0.0001,0.75,5,96,55,55,Layer1_pool_GPU,0,0);
	dim3 Norm11_Block(96,1,1);
	dim3 Norm11_Thread(32,23);   
	executelrnNormCuda_split<<<Norm11_Block,Norm11_Thread>>>(Layer1_Norm_GPU,0.0001,0.75,5,96,55,55,Layer1_pool_GPU,0,32);
	dim3 Norm12_Block(96,1,1);
	dim3 Norm12_Thread(23,32);   
	executelrnNormCuda_split<<<Norm12_Block,Norm12_Thread>>>(Layer1_Norm_GPU,0.0001,0.75,5,96,55,55,Layer1_pool_GPU,32,0);
	dim3 Norm13_Block(96,1,1);
	dim3 Norm13_Thread(23,23);   
	executelrnNormCuda_split<<<Norm13_Block,Norm13_Thread>>>(Layer1_Norm_GPU,0.0001,0.75,5,96,55,55,Layer1_pool_GPU,32,32);
	/* Max Pool */
	hipMalloc((void**) &Layer2_Neurons_GPU,sizeof(float)*L1_OUT * POOL1_FMAP);
	dim3 pool1_Block(96,1,1);
	dim3 pool1_Thread(27,27);   
	executepoolingCuda<<<pool1_Block,pool1_Thread>>>(Layer1_pool_GPU,Layer2_Neurons_GPU,96,27,27,3,2,55,55);

	/* Second Layer convolution + ReLU + pooling */ 
	float *Layer2_bias_GPU,*Layer2_Weights_GPU,*Layer2_Norm_GPU,*Layer2_pool_GPU,*Layer3_Neurons_GPU;

	hipMalloc((void**) &Layer2_Weights_GPU,sizeof(float)*(L2_KERNEL_SIZE * L2_OUT));
	hipMalloc((void**) &Layer2_bias_GPU, sizeof(float)* L2_OUT);
	hipMalloc((void**) &Layer2_Norm_GPU, sizeof(float)* L2_OUT * L2_FMAP);

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer2_Weights_GPU,Layer2_Weights_CPU, sizeof(float)*(L2_KERNEL_SIZE * L2_OUT), hipMemcpyHostToDevice);
	hipMemcpy(Layer2_bias_GPU,bias_2, sizeof(float)* L2_OUT,hipMemcpyHostToDevice);
 
    /* Group = 2 and each group is 128*27*27 */
	dim3 Layer2_Block(128,1,1);
	dim3 Layer2_Thread(27,27);   
	execute3DconvolutionCuda<<<Layer2_Block,Layer2_Thread>>>(Layer2_bias_GPU,Layer2_Neurons_GPU,Layer2_Weights_GPU,Layer2_Norm_GPU,128,27,27,1,5,2,48,2);
	execute3Dconvolutiongroup2Cuda<<<Layer2_Block,Layer2_Thread>>>(Layer2_bias_GPU,Layer2_Neurons_GPU,Layer2_Weights_GPU,Layer2_Norm_GPU,128,27,27,1,5,2,48,2);

	/*Normalisation */	
	(hipMalloc((void**) &Layer2_pool_GPU,sizeof(float)*L2_OUT * L2_FMAP));
	dim3 Norm2_Block(256,1,1);
	dim3 Norm2_Thread(27,27);   
	executelrnNormCuda<<<Norm2_Block,Norm2_Thread>>>(Layer2_Norm_GPU,0.0001,0.75,5,256,27,27,Layer2_pool_GPU,0);
	/* Max Pool */
	hipMalloc((void**) &Layer3_Neurons_GPU,sizeof(float)*L2_OUT * POOL2_FMAP);
	dim3 pool2_Block(256,1,1);
	dim3 pool2_Thread(13,13);   
	executepoolingCuda<<<pool2_Block,pool2_Thread>>>(Layer2_pool_GPU,Layer3_Neurons_GPU,256,13,13,3,2,27,27);

	/* Third Layer convolution + ReLU  */ 
	float *Layer3_bias_GPU,*Layer3_Weights_GPU,*Layer4_Neurons_GPU;
	hipMalloc((void**) &Layer3_Weights_GPU,sizeof(float)*(L3_KERNEL_SIZE * L3_OUT));
	hipMalloc((void**) &Layer3_bias_GPU, sizeof(float)*L3_OUT);
	hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(float)*(L3_FMAP * L3_OUT));

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer3_Weights_GPU,Layer3_Weights_CPU, sizeof(float)*(L3_KERNEL_SIZE * L3_OUT), hipMemcpyHostToDevice);
	hipMemcpy(Layer3_bias_GPU,bias_3, sizeof(float)*L3_OUT,hipMemcpyHostToDevice);
	dim3 Layer3_Block(384,1,1);
	dim3 Layer3_Thread(13,13);   
	execute3DconvolutionCuda<<<Layer3_Block,Layer3_Thread>>>(Layer3_bias_GPU,Layer3_Neurons_GPU,Layer3_Weights_GPU,Layer4_Neurons_GPU,384,13,13,1,3,1,256,1);

	/* Fourth Layer convolution + ReLU  */
	float *Layer4_bias_GPU,*Layer4_Weights_GPU,*Layer5_Neurons_GPU;
	hipMalloc((void**) &Layer4_Weights_GPU,sizeof(float)*(L4_KERNEL_SIZE * L4_OUT));
	hipMalloc((void**) &Layer4_bias_GPU, sizeof(float)*L4_OUT);
	hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(float)*(L4_FMAP * L4_OUT));

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer4_Weights_GPU,Layer4_Weights_CPU, sizeof(float)*(L4_KERNEL_SIZE * L4_OUT), hipMemcpyHostToDevice);
	hipMemcpy(Layer4_bias_GPU,bias_4, sizeof(float)*L4_OUT,hipMemcpyHostToDevice);
	dim3 Layer4_Block(192,1,1);
	dim3 Layer4_Thread(13,13);   
	execute3DconvolutionCuda<<<Layer4_Block,Layer4_Thread>>>(Layer4_bias_GPU,Layer4_Neurons_GPU,Layer4_Weights_GPU,Layer5_Neurons_GPU,192,13,13,1,3,1,192,2);
	execute3Dconvolutiongroup2Cuda<<<Layer4_Block,Layer4_Thread>>>(Layer4_bias_GPU,Layer4_Neurons_GPU,Layer4_Weights_GPU,Layer5_Neurons_GPU,192,13,13,1,3,1,192,2);

	/* Fifth Layer convolution + ReLU + pooling */
	float *Layer5_bias_GPU,*Layer5_Weights_GPU,*Layer5_pool_GPU,*Layer6_Neurons_GPU;
	hipMalloc((void**) &Layer5_Weights_GPU,sizeof(float)*(L5_KERNEL_SIZE * L5_OUT));
	hipMalloc((void**) &Layer5_bias_GPU, sizeof(float)*L5_OUT);
	hipMalloc((void**) &Layer5_pool_GPU, sizeof(float)*(L5_FMAP * L5_OUT));

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer5_Weights_GPU,Layer5_Weights_CPU, sizeof(float)*(L5_KERNEL_SIZE * L5_OUT), hipMemcpyHostToDevice);
	hipMemcpy(Layer5_bias_GPU,bias_5, sizeof(float)*L5_OUT,hipMemcpyHostToDevice);
	dim3 Layer5_Block(128,1,1);
	dim3 Layer5_Thread(13,13);   
	execute3DconvolutionCuda<<<Layer5_Block,Layer5_Thread>>>(Layer5_bias_GPU,Layer5_Neurons_GPU,Layer5_Weights_GPU,Layer5_pool_GPU,128,13,13,1,3,1,192,2);
	execute3Dconvolutiongroup2Cuda<<<Layer5_Block,Layer5_Thread>>>(Layer5_bias_GPU,Layer5_Neurons_GPU,Layer5_Weights_GPU,Layer5_pool_GPU,128,13,13,1,3,1,192,2);
	hipMalloc((void**) &Layer6_Neurons_GPU,sizeof(float)*L5_OUT * POOL3_FMAP);
	dim3 pool5_Block(256,1,1);
	dim3 pool5_Thread(6,6);   
	executepoolingCuda<<<pool5_Block,pool5_Thread>>>(Layer5_pool_GPU,Layer6_Neurons_GPU,256,6,6,3,2,13,13);

	/* Sixth Layer Fully connected + ReLU */	
	float *Layer6_bias_GPU; 
	float *Layer6_Weights_GPU;
	float *Layer7_Neurons_GPU;

	hipMalloc((void**) &Layer6_Weights_GPU,sizeof(float)*4096*256*6*6);
	hipMalloc((void**) &Layer6_bias_GPU, sizeof(float)*4096);
	hipMalloc((void**) &Layer7_Neurons_GPU, sizeof(float)*4096);

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer6_Weights_GPU,Layer6_Weights_CPU, sizeof(float)*4096*256*6*6, hipMemcpyHostToDevice);
	hipMemcpy(Layer6_bias_GPU,bias_6, sizeof(float)*4096,hipMemcpyHostToDevice);

	dim3 Layer6_Block(4096,1,1);
	dim3 Layer6_Thread(1,1);   // combi tried 10*10*10
	executeFCLayer<<<Layer6_Block,Layer6_Thread>>>(Layer6_bias_GPU,Layer6_Neurons_GPU,Layer6_Weights_GPU,Layer7_Neurons_GPU,4096,(256*6*6),true,false);
	// RELU LAyer 

	/* Seventh Layer Fully connected + ReLU */	
	float *Layer7_bias_GPU; 
	float *Layer7_Weights_GPU;
	float *Layer8_Neurons_GPU;

	hipMalloc((void**) &Layer7_Weights_GPU,sizeof(float)*4096*4096);
	hipMalloc((void**) &Layer7_bias_GPU, sizeof(float)*4096);
	hipMalloc((void**) &Layer8_Neurons_GPU, sizeof(float)*4096);

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer7_Weights_GPU,Layer7_Weights_CPU, sizeof(float)*4096*4096, hipMemcpyHostToDevice);
	hipMemcpy(Layer7_bias_GPU,bias_7, sizeof(float)*4096,hipMemcpyHostToDevice);

	dim3 Layer7_Block(4096,1,1);
	dim3 Layer7_Thread(1,1);   // combi tried 10*10*10
	executeFCLayer<<<Layer7_Block,Layer7_Thread>>>(Layer7_bias_GPU,Layer7_Neurons_GPU,Layer7_Weights_GPU,Layer8_Neurons_GPU,4096,4096,true,false);

	/* Eigth Layer Fully connected + ReLU */	
	float *Layer8_bias_GPU; 
	float *Layer9_Neurons_GPU;
	float *Layer8_Weights_GPU;

	hipMalloc((void**) &Layer8_Weights_GPU,sizeof(float)*4096*1000);
	hipMalloc((void**) &Layer8_bias_GPU, sizeof(float)*1000);
	hipMalloc((void**) &Layer9_Neurons_GPU, sizeof(float)*1000);

	/* Memcpy of weights and bias */ 
	hipMemcpy(Layer8_Weights_GPU,Layer8_Weights_CPU, sizeof(float)*4096*1000, hipMemcpyHostToDevice);
	hipMemcpy(Layer8_bias_GPU,bias_8, sizeof(float)*1000,hipMemcpyHostToDevice);

	dim3 Layer8_Block(1000,1,1);
	dim3 Layer8_Thread(1,1);   // combi tried 10*10*10
	executeFCLayer<<<Layer8_Block,Layer8_Thread>>>(Layer8_bias_GPU,Layer8_Neurons_GPU,Layer8_Weights_GPU,Layer9_Neurons_GPU,1000,4096,false,false);

	float *fc9_Neurons_CPU = (float *)malloc(sizeof(float) * (1000));
	hipMemcpy(fc9_Neurons_CPU,Layer9_Neurons_GPU, sizeof(float)*(1000), hipMemcpyDeviceToHost);
	/* Check the output */
	float max = 0.0;int index = 0; 
	for(int i =0; i < 1000; i++)
	{
		if(max < fc9_Neurons_CPU[i])
		{
			max = fc9_Neurons_CPU[i];
			index = i;
		}
	}
	printf("INDEX = %d\n",index);
    hipFree(Layer1_Neurons_GPU);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer1_bias_GPU);
    hipFree(Layer1_Norm_GPU);
    hipFree(Layer1_pool_GPU);
    hipFree(Layer2_Neurons_GPU);
    hipFree(Layer2_Weights_GPU);
    hipFree(Layer2_bias_GPU);
    hipFree(Layer2_pool_GPU);
    hipFree(Layer2_Norm_GPU);
    hipFree(Layer3_Neurons_GPU);
    hipFree(Layer3_Weights_GPU);
    hipFree(Layer3_bias_GPU);
    hipFree(Layer4_Neurons_GPU);
    hipFree(Layer4_Weights_GPU);
    hipFree(Layer4_bias_GPU);
    hipFree(Layer5_Neurons_GPU);
    hipFree(Layer5_Weights_GPU);
    hipFree(Layer5_bias_GPU);
    hipFree(Layer5_pool_GPU);
    hipFree(Layer6_Neurons_GPU);
    hipFree(Layer6_Weights_GPU);
    hipFree(Layer6_bias_GPU);
    hipFree(Layer7_Neurons_GPU);
    hipFree(Layer7_bias_GPU);
    hipFree(Layer7_Weights_GPU);
    hipFree(Layer8_Neurons_GPU);
    hipFree(Layer8_Weights_GPU);
    hipFree(Layer8_bias_GPU);
    hipFree(Layer9_Neurons_GPU);
    free(Layer1_Neurons_CPU);
    free(Layer1_Weights_CPU);
    free(Layer2_Weights_CPU);
    free(Layer3_Weights_CPU);
    free(Layer4_Weights_CPU);
    free(Layer5_Weights_CPU);
    free(Layer6_Weights_CPU);
    free(Layer7_Weights_CPU);
    free(Layer8_Weights_CPU);
#endif
	/* SoftMax */
	//Confirm the functionality of SoftMax ,extract_weights("data/fc8_out.txt",fc9_Neurons_CPU,false);
	//executeSoftMax(fc9_Neurons_CPU);
	exit(0);
}


			
